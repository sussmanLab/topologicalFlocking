#include "hip/hip_runtime.h"
#include "hyperrectangularCellList.cuh"
#include "indexer.h"
#include "periodicBoundaryConditions.h"
/*! \file hyperrectangularCellList.cu */

/*!
    \addtogroup utilityKernels
    @{
*/

/*!
  Assign particles to bins, keep track of the number of particles per bin, etc.
  */
__global__ void gpu_compute_cell_list_kernel(dVec *d_pt,
                                              unsigned int *d_elementsPerCell,
                                              int *d_particleIndices,
                                              dVec *d_cellParticlePos,
                                              int Np,
                                              unsigned int Nmax,
                                              iVec gridCellsPerSide,
                                              dVec gridCellSizes,
                                              IndexDD cellIndexer,
                                              Index2D cellListIndexer,
                                              int *d_assist
                                              )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= Np)
        return;

    dVec pos = d_pt[idx];
    iVec bin;
    for (int dd = 0; dd < DIMENSION; ++dd)
        bin.x[dd] = floor(pos.x[dd] / gridCellSizes.x[dd]);
    int binIndex = cellIndexer(bin);
    unsigned int offset = atomicAdd(&(d_elementsPerCell[binIndex]), 1);
    if(offset < Nmax)
        {
        unsigned int write_pos = min(cellListIndexer(offset,binIndex),cellListIndexer.getNumElements()-1);
        d_particleIndices[write_pos] = idx;
        d_cellParticlePos[write_pos] = pos;
        }
    else
        {
        d_assist[0] = offset+1;
        d_assist[1] = 1;
        };
    };

bool gpu_compute_cell_list(dVec *d_pt,
                                  unsigned int *d_cell_sizes,
                                  int *d_idx,
                                  dVec *d_cellParticlePos,
                                  int Np,
                                  int &Nmax,
                                  iVec gridCellsPerSide,
                                  dVec gridCellSizes,
                                  BoxPtr Box,
                                  IndexDD &ci,
                                  Index2D &cli,
                                  int *d_assist
                                  )
    {
    //optimize block size later
    unsigned int block_size = 128;
    if (Np < 128) block_size = 16;
    unsigned int nblocks  = Np/block_size + 1;


    unsigned int nmax = (unsigned int) Nmax;
    gpu_compute_cell_list_kernel<<<nblocks, block_size>>>(d_pt,
                                                          d_cell_sizes,
                                                          d_idx,
                                                          d_cellParticlePos,
                                                          Np,
                                                          nmax,
                                                          gridCellsPerSide,
                                                          gridCellSizes,
                                                          ci,
                                                          cli,
                                                          d_assist
                                                          );
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    }

/** @} */ //end of group declaration
